#include "hip/hip_runtime.h"
#include "adagrad_op.h"
#include "caffe2/core/common_gpu.h"
#include "caffe2/core/context_gpu.h"

namespace caffe2 {

__global__ void AdagradUpdate(
    int N,
    const float* g,
    const float* h,
    float* ng,
    float* nh,
    float epsilon,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = nh[i] = h[i] + gi * gi;
    ng[i] = lr[0] * gi / (std::sqrt(hi) + epsilon);
  }
}

template <>
void adagrad_update<HIPContext>(
    int N,
    const float* g,
    const float* h,
    float* ng,
    float* nh,
    float epsilon,
    const float* lr,
    HIPContext* context) {
  AdagradUpdate<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, g, h, ng, nh, epsilon, lr);
}

__global__ void AdagradCompute(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    const float* lr) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = nh[i] = h[i] + gi * gi;
    nw[i] = w[i] + lr[0] * gi / (std::sqrt(hi) + epsilon);
  }
}

template <>
void adagrad_compute<HIPContext>(
    int N,
    const float* w,
    const float* g,
    const float* h,
    float* nw,
    float* nh,
    float epsilon,
    const float* lr,
    HIPContext* context) {
  AdagradCompute<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, w, g, h, nw, nh, epsilon, lr);
}

namespace {
REGISTER_HIP_OPERATOR(Adagrad, AdagradOp<float, HIPContext>);
}
}
