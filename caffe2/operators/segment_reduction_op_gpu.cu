#include "caffe2/core/context_gpu.h"
#include "caffe2/core/operator.h"
#include "caffe2/utils/math.h"

namespace caffe2 {

template <
    typename T,
    class Context = HIPContext,
    bool FIRSTDIMS = true,
    bool NORMALIZE = false>
class ReduceDimsOp : public Operator<HIPContext> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  ReduceDimsOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws),
        num_reduce_dims_(
            OperatorBase::GetSingleArgument<int32_t>("num_reduce_dim", 1)) {}

  ~ReduceDimsOp() {}

  bool RunOnDevice() override {
    const auto& input = Input(0);
    const auto* input_data = input.template data<T>();
    auto* Y = Output(0);

    CHECK_LE(num_reduce_dims_, input.dims().size());
    const int M = FIRSTDIMS
        ? input.size_to_dim(num_reduce_dims_)
        : input.size_to_dim(input.ndim() - num_reduce_dims_);
    const int N = FIRSTDIMS
        ? input.size_from_dim(num_reduce_dims_)
        : input.size_from_dim(input.ndim() - num_reduce_dims_);

    vector<TIndex> output_shape;
    int start_index = FIRSTDIMS ? num_reduce_dims_ : 0;
    int end_index = FIRSTDIMS ? input.dims().size()
                              : input.dims().size() - num_reduce_dims_;
    for (int i = start_index; i < end_index; ++i) {
      output_shape.push_back(input.dims()[i]);
    }

    Y->Resize(output_shape);

    int in_dim = FIRSTDIMS ? M : N;

    if (ones_.size() != in_dim) {
      ones_.Resize(in_dim);
      math::Set<T, Context>(
          in_dim,
          static_cast<T>(1),
          ones_.template mutable_data<T>(),
          &context_);
    }

    T alpha = 1.0;
    if (NORMALIZE) { // Static if
      alpha = 1.0 / in_dim;
    }

    math::Gemv<T, Context>(
        FIRSTDIMS ? CblasTrans : CblasNoTrans,
        M,
        N,
        alpha,
        input_data,
        ones_.template data<T>(),
        0.0,
        Y->template mutable_data<T>(),
        &context_);

    return true;
  }

 private:
  Tensor<Context> ones_;
  int num_reduce_dims_;
};

template <
    typename T,
    class Context = HIPContext,
    bool FIRSTDIMS = true,
    bool NORMALIZE = false>
class ReduceDimsGradientOp : public Operator<HIPContext> {
 public:
  USE_OPERATOR_CONTEXT_FUNCTIONS;
  ReduceDimsGradientOp(const OperatorDef& operator_def, Workspace* ws)
      : Operator<HIPContext>(operator_def, ws),
        num_reduce_dims_(
            OperatorBase::GetSingleArgument<int32_t>("num_reduce_dim", 1)) {}

  ~ReduceDimsGradientOp() {}

  bool RunOnDevice() override {
    const auto& grad_in = Input(0);
    const auto& in_shape = Input(1);

    Tensor<CPUContext> shape;
    shape.CopyFrom(in_shape);
    // Copy first dims
    vector<TIndex> output_shape(
        shape.template data<TIndex>(),
        shape.template data<TIndex>() + shape.size());

    auto* out_grad = Output(0);
    out_grad->Resize(output_shape);

    const int M = FIRSTDIMS
        ? out_grad->size_to_dim(num_reduce_dims_)
        : out_grad->size_to_dim(out_grad->ndim() - num_reduce_dims_);
    const int N = FIRSTDIMS
        ? out_grad->size_from_dim(num_reduce_dims_)
        : out_grad->size_from_dim(out_grad->ndim() - num_reduce_dims_);

    int in_dim = FIRSTDIMS ? M : N;

    T alpha = 1.0;
    if (NORMALIZE) { // Static if
      alpha = 1.0 / in_dim;
    }

    math::Set<T, HIPContext>(
        out_grad->size(),
        FIRSTDIMS ? static_cast<T>(0) : static_cast<T>(alpha),
        out_grad->template mutable_data<T>(),
        &context_);

    for (int i = 0; i < M; ++i) {
      if (FIRSTDIMS) {
        math::Axpby<T, HIPContext>(
            N,
            alpha,
            grad_in.template data<T>(),
            static_cast<T>(0),
            out_grad->template mutable_data<T>() + i * N,
            &context_);
      } else {
        math::Scale<T, HIPContext>(
            N,
            grad_in.template data<T>() + i,
            out_grad->template data<T>() + i * N,
            out_grad->template mutable_data<T>() + i * N,
            &context_);
      }
    }

    return true;
  }

 private:
  int num_reduce_dims_;
};

REGISTER_HIP_OPERATOR(ReduceFrontSum, ReduceDimsOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    ReduceFrontSumGradient,
    ReduceDimsGradientOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    ReduceFrontMean,
    ReduceDimsOp<float, HIPContext, true, true>);
REGISTER_HIP_OPERATOR(
    ReduceFrontMeanGradient,
    ReduceDimsGradientOp<float, HIPContext, true, true>);

REGISTER_HIP_OPERATOR(ReduceBackSum, ReduceDimsOp<float, HIPContext, false>);
REGISTER_HIP_OPERATOR(
    ReduceBackSumGradient,
    ReduceDimsGradientOp<float, HIPContext, false, false>);

REGISTER_HIP_OPERATOR(
    ReduceBackMean,
    ReduceDimsOp<float, HIPContext, false, true>);
REGISTER_HIP_OPERATOR(
    ReduceBackMeanGradient,
    ReduceDimsGradientOp<float, HIPContext, false, true>);

} // namespace caffe2
