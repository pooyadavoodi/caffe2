#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "caffe2/operators/recurrent_network_op.h"

namespace caffe2 {

namespace detail {

template <typename T, typename Context>
void initializeRecurrentInput(
    const RecurrentInput& rc,
    int32_t seqLen,
    int32_t batchSize,
    Workspace* ws,
    Context* context);

namespace {

template <typename T>
__global__
void initRecurrentInput_kernel(
    size_t stateSize,
    const T* input,
    T* state) {
  // index into appropriate target buffer
  const int block_id = blockIdx.x;
  T* state_local = state + block_id*stateSize;

  // copy
  for (int idx=threadIdx.x; idx < stateSize; idx+=blockDim.x) {
    state_local[idx] = input[idx];
  }
}


}; // namespace

template <>
void initializeRecurrentInput<float,HIPContext>(
    const RecurrentInput& rc,
    int32_t seqLen,
    int32_t batchSize,
    Workspace* ws,
    HIPContext* context) {
  auto stateBlob = ws->GetBlob(rc.state);
  CAFFE_ENFORCE(stateBlob);
  auto* state = stateBlob->GetMutable<Tensor<HIPContext>>();

  auto inputBlob = ws->GetBlob(rc.input);
  CAFFE_ENFORCE(inputBlob);
  const auto& input = inputBlob->Get<Tensor<HIPContext>>();
  CAFFE_ENFORCE_GE(input.ndim(), 1, rc.input);
  CAFFE_ENFORCE_LE(input.ndim(), 3, rc.input);

  const auto stateSize = input.dim(input.ndim() - 1);
  // States at [0, ..., T] (inclusive)
  state->Resize(seqLen + 1, batchSize, stateSize);

  if (input.ndim() == 3) {
    CAFFE_ENFORCE_EQ(input.dim(0), 1, rc.input);
  }
  if (input.ndim() >= 2) {
    CAFFE_ENFORCE_EQ(input.dim(input.ndim() - 2), batchSize, rc.input);
    context->Copy<float, HIPContext, HIPContext>(
        batchSize * stateSize,
        input.data<float>(),
        state->mutable_data<float>());
  } else {
    initRecurrentInput_kernel<float><<<batchSize, 128, 0, context->hip_stream()>>>(
        stateSize,
        input.data<float>(),
        state->mutable_data<float>());
  }
}

}; // namespace detail

namespace {
REGISTER_HIP_OPERATOR(
    RecurrentNetwork,
    RecurrentNetworkOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(
    RecurrentNetworkGradient,
    RecurrentNetworkGradientOp<float, HIPContext>);
}; // namespace

} // namespace caffe2
