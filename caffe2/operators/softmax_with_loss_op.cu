#include "hip/hip_runtime.h"
#include <cfloat>

#include "caffe2/core/context_gpu.h"
#include "softmax_with_loss_op.h"

namespace caffe2 {

namespace {

__global__ void LabelCrossEntropyKernel(
    const int N, const int D, const float* Pdata, const int* labeldata,
    const float* weights, float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    CUDA_KERNEL_ASSERT(labeldata[i] >= 0 && labeldata[i] < D);
    float weight = weights ? weights[i] : 1.0;
    Ydata[i] = -logf(max(Pdata[i * D + labeldata[i]], FLT_MIN)) * weight;
  }
}

__global__ void LabelCrossEntropyGradientKernel(
    const int N,
    const int D,
    const float* Pdata,
    const int* labeldata,
    float* dXdata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    int idx = i * D + labeldata[i];
    dXdata[idx] = Pdata[idx] - 1.;
  }
}

__global__ void LabelCrossEntropyGradientKernelWeighted(
    const int N,
    const int D,
    const float* Pdata,
    const int* labeldata,
    float* dXdata,
    const float* weights) {
  CUDA_1D_KERNEL_LOOP(i, N * D) {
    int row = i / D;
    int d = i % D;
    float val = Pdata[i] - 1.0 * (d == labeldata[row]);
    float weight = weights[row];
    dXdata[i] = val * weight;
  }
}

__global__ void ProbCrossEntropyKernel(
    const int N,
    const int D,
    const float* Pdata,
    const float* labeldata,
    const float* weights,
    float* Ydata) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    float weight = weights ? weights[i] : 1.0;
    float total_prob = 0.0;
    Ydata[i] = 0.0;
    for (int j = 0; j < D; j++) {
      int idx = i * D + j;
      total_prob += labeldata[idx];
      CUDA_KERNEL_ASSERT(labeldata[idx] >= 0);
      Ydata[i] += -logf(max(Pdata[idx], FLT_MIN)) * labeldata[idx] * weight;
    }
    CUDA_KERNEL_ASSERT(abs(total_prob - 1.0) < 1e-5f);
  }
}

__global__ void ProbCrossEntropyGradientKernel(
    const int N,
    const int D,
    const float* Pdata,
    const float* labeldata,
    float* dXdata,
    const float* weights) {
  if (weights == NULL) {
    CUDA_1D_KERNEL_LOOP(i, N) {
      for (int j = 0; j < D; j++) {
        int idx = i * D + j;
        dXdata[idx] = Pdata[idx] - labeldata[idx];
      }
    }
  } else {
    CUDA_1D_KERNEL_LOOP(i, N) {
      float weight = weights[i];
      for (int d = 0; d < D; d++) {
        int idx = i * D + d;
        dXdata[idx] = (Pdata[idx] - labeldata[idx]) * weight;
      }
    }
  }
}

#define REDUCTION_KERNEL_THREADS_X 128
#define REDUCTION_KERNEL_THREADS_Y 4
#define REDUCTION_THREADS \
  (REDUCTION_KERNEL_THREADS_X * REDUCTION_KERNEL_THREADS_Y)

__global__ void
RowMaxKernelLargeD(const int num, const int D, const float* data, float* out) {
  __shared__ float
      max_buffer[REDUCTION_KERNEL_THREADS_Y * REDUCTION_KERNEL_THREADS_X];
  const int threadId = threadIdx.y * REDUCTION_KERNEL_THREADS_X + threadIdx.x;

  for (int index = blockIdx.y * blockDim.y + threadIdx.y; index < num;
       index += blockDim.y * gridDim.y) {
    float maxval = -FLT_MAX;
    for (int x = blockIdx.x * blockDim.x + threadIdx.x; x < D;
         x += blockDim.x * gridDim.x) {
      maxval = fmaxf(data[index * D + x], maxval);
    }
    max_buffer[threadId] = maxval;

    __syncthreads();

    if (threadIdx.x < 32) {
      maxval = fmaxf(
          fmaxf(
              fmaxf(maxval, max_buffer[threadId + 32]),
              max_buffer[threadId + 64]),
          max_buffer[threadId + 96]);
      max_buffer[threadId] = maxval;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
#pragma unroll
      for (int j = 1; j < 32; j++) {
        maxval = max(max_buffer[threadId + j], maxval);
      }
      out[index] = maxval;
    }
    __syncthreads();
  }
}

__global__ void RowMaxKernel(const int num, const int D, const float* data,
    float* out) {
  CUDA_1D_KERNEL_LOOP(index, num) {
    float maxval = -FLT_MAX;
    for (int d = 0; d < D; ++d) {
      maxval = max(data[index * D + d], maxval);
    }
    out[index] = maxval;
  }
}

__global__ void SpatialSoftmaxKernel(const int num, const int D, const int W, const int H,
      const float* Xdata, float* Pdata) {
  CUDA_1D_KERNEL_LOOP(index, num * W * H) {
    int x = index % W;
    int y = (index / W) % H;
    int i = index / W / H;

    // Subtract max on each cell for numerical reasons
    float max_val = -FLT_MAX;
    for(int c = 0; c < D; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      max_val = max(max_val, Xdata[idx]);
    }

    // Exponentiate
    float expsum = 0.0f;
    for(int c = 0; c < D; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      float expx = exp(Xdata[idx] - max_val);
      Pdata[idx] = expx;
      expsum += expx;
    }

    // Normalize
    for(int c=0; c<D; ++c) {
      int idx = i * (H * W * D) + c * (H * W) + y * W + x;
      Pdata[idx] /= expsum;
    }
  }
}


#define DONTCARE (-1)

__global__ void SpatialCrossEntropyLossKernel(const int N, const int D, const int W, const int H,
    const float* Pdata, const int* label_data, const float *weights,
      float* loss_data, float* weight_data) {
  CUDA_1D_KERNEL_LOOP(index, N * W * H) {
    int x = index % W;
    int y = (index / W) % H;
    int i = index / W / H;
    const int label = static_cast<int>(label_data[index]);

    if (label != DONTCARE) {
      CUDA_KERNEL_ASSERT(label >= 0 && label < D);
      float weight = (weights == NULL ? 1.0 : weights[index]);
      loss_data[index] = -log(max(
        Pdata[i * W * H * D + label * W * H + y * W + x], 1e-20f)) * weight;
      weight_data[index] = weight;
    } else {
      loss_data[index] = 0;
      weight_data[index] = 0;
    }
  }
}

__global__ void SpatialSoftmaxLossGradientKernel(const int N, const int D,
    const int W, const int H, const int* label_data, const float* weights,
         float* dX_data, float* weights_) {
 CUDA_1D_KERNEL_LOOP(index, N * W * H) {
   int x = index % W;
   int y = (index / W) % H;
   int i = index / W / H;
   const int label = static_cast<int>(label_data[index]);

   if (label != DONTCARE) {
     int data_idx = i * (H * W * D) + label * (H * W) + y * W + x;
     dX_data[data_idx] -= 1.0;
     if (weights != NULL) {
       float weight = weights[index];
       for (int c = 0; c < D; ++c) {
         int data_idx = i * (H * W * D) + c * (H * W) + y * W + x;
         dX_data[data_idx] *= weight;
       }
       weights_[index] = weight;
     } else {
       weights_[index] = 1.0;
     }
   } else {
     // Ignore-label, so set all gradients for this positions
     // tp zero
     for (int c = 0; c < D; ++c) {
       int data_idx = i * (H * W * D) + c * (H * W) + y * W + x;
       dX_data[data_idx] = 0.0;
     }
     weights_[index] = 0.0;
   }
 }
}

__global__ void SoftmaxNormalizeKernel(
    const int nthreads, const int D, const float* Pdata, const float* scales,
    float* out) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    int n = index / D;
    out[index] = Pdata[index] / scales[n];
  }
}

void Softmax(
    const int N,
    const int D,
    const float* logits,
    const float* sum_multiplier,
    float* scales,
    float* probs,
    HIPContext* context) {
  const int size = N * D;

  if (D > 512) {
    dim3 threadsPerBlock(
        REDUCTION_KERNEL_THREADS_X, REDUCTION_KERNEL_THREADS_Y);
    dim3 numBlocks(1, max(1, N / 32));
    RowMaxKernelLargeD<<<
        numBlocks,
        threadsPerBlock,
        0,
        context->hip_stream()>>>(N, D, logits, scales);
  } else {
    RowMaxKernel<<<
        CAFFE_GET_BLOCKS(N),
        CAFFE_CUDA_NUM_THREADS,
        0,
        context->hip_stream()>>>(N, D, logits, scales);
  }
  // Put the intermediate result X - max(X) into Y
  context->Copy<float, HIPContext, HIPContext>(size, logits, probs);
  // Subtract the scale
  math::Gemm<float, HIPContext>(CblasNoTrans, CblasNoTrans, N, D, 1,
                                 -1, scales, sum_multiplier, 1, probs, context);
  // Exponentiation
  math::Exp<float, HIPContext>(size, probs, probs, context);
  // Sum exponentiated values
  math::Gemv<float, HIPContext>(CblasNoTrans, N, D, 1, probs, sum_multiplier,
                                 0, scales, context);
  // Normalize
  SoftmaxNormalizeKernel<<<CAFFE_GET_BLOCKS(size), CAFFE_CUDA_NUM_THREADS,
                           0, context->hip_stream()>>>(
    size, D, probs, scales, probs);
}

} // namespace

template<>
bool SoftmaxWithLossOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Logits
  auto& T = Input(1);  // Labels / targets
  auto* P = Output(0); // Probabilities from softmax
  auto* avg_loss = Output(1); // Average loss
  const float* weights = (InputSize() > 2 ? Input(2).data<float>() : NULL);

  int N = X.dim32(0);
  int D = X.dim32(1);
  P->ResizeLike(X);
  total_weight_ptr_.Resize(1);
  DCHECK(!(spatial_mode_ && label_prob_mode_)); // Do not currently support both
  if (!spatial_mode_) {
    DCHECK_EQ(X.ndim(), 2);
    if (!label_prob_mode_) {
      DCHECK((T.ndim() == 1) || (T.ndim() == 2 && T.dim32(1) == 1));
    } else {
      DCHECK(T.ndim() == 2 && T.dim32(0) == N && T.dim32(1) == D);
    }
    DCHECK_EQ(T.dim32(0), N);

    avg_loss->Resize(vector<TIndex>());
    if (losses_.size() != N) {
      losses_.Resize(N);
    }
    if (sum_multiplier_.size() != D) {
      sum_multiplier_.Resize(D);
      math::Set<float, HIPContext>(
          D, 1.f, sum_multiplier_.mutable_data<float>(), &context_);
    }
    Softmax(
        N,
        D,
        X.data<float>(),
        sum_multiplier_.data<float>(),
        losses_.mutable_data<float>(),
        P->mutable_data<float>(),
        &context_);
    // Compute label xent loss per example
    if (!label_prob_mode_) {
      LabelCrossEntropyKernel<<<
          CAFFE_GET_BLOCKS(N),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          N,
          D,
          P->data<float>(),
          T.data<int>(),
          weights,
          losses_.mutable_data<float>());
    } else {
      ProbCrossEntropyKernel<<<
          CAFFE_GET_BLOCKS(N),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          N,
          D,
          P->data<float>(),
          T.data<float>(),
          weights,
          losses_.mutable_data<float>());
    }

    float total_weight = N;
    if (weights) {
      // Sum weights
      math::Sum<float, HIPContext>(N, weights,
        total_weight_ptr_.mutable_data<float>(), &context_);
      hipMemcpyAsync(&total_weight, total_weight_ptr_.data<float>(),
        sizeof(float), hipMemcpyDeviceToHost, context_.hip_stream());
    }

    // Sum of all losses
    float* avg_loss_data = avg_loss->mutable_data<float>();
    math::Sum<float, HIPContext>(
        losses_.size(), losses_.data<float>(), avg_loss_data, &context_);
    // Average of input batch size
    if (total_weight > 0) {
      math::Scale<float, HIPContext>(
          1, scale_ / total_weight, avg_loss_data, avg_loss_data, &context_);
    }
  } else {
    DCHECK_EQ(X.ndim(), 4);
    DCHECK_EQ(T.ndim(), 3);
    DCHECK_EQ(T.dim32(0), N);

    int H = X.dim32(2);
    int W = X.dim32(3);
    if (losses_.size() != N * W * H) {
      losses_.Resize(N * W * H);
    }
    if (weights_.size() != N * W * H) {
      weights_.Resize(N * W * H);
    }

    const float* Xdata = X.data<float>();
    float* Pdata = P->mutable_data<float>();

    // Softmax for each x,y location
    SpatialSoftmaxKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                           0, context_.hip_stream()>>>(
        N, D, W, H, Xdata, Pdata);

    // Cross entropy
    avg_loss->Resize(vector<TIndex>());
    float* avg_loss_data = avg_loss->mutable_data<float>();
    math::Set<float, HIPContext>(1, 0.0f, avg_loss_data, &context_);

    const int* label_data = T.data<int>();
    math::Set<float, HIPContext>(
      1, 0.0f, total_weight_ptr_.mutable_data<float>(), &context_);

    SpatialCrossEntropyLossKernel<<<CAFFE_GET_BLOCKS(N * W * H),
      CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
        N, D, W, H, P->data<float>(), label_data, weights,
        losses_.mutable_data<float>(), weights_.mutable_data<float>());


    // Somewhat awkward scalar passing from device to host
    float h_total_weight;
    math::Sum<float, HIPContext>(
      weights_.size(), weights_.data<float>(),
      total_weight_ptr_.mutable_data<float>(), &context_);
    hipMemcpyAsync(&h_total_weight, total_weight_ptr_.data<float>(),
      sizeof(float), hipMemcpyDeviceToHost, context_.hip_stream());

    math::Sum<float, HIPContext>(
        losses_.size(), losses_.data<float>(), avg_loss_data, &context_);

    // Final scaling
    if (h_total_weight > 0) {
      math::Scale<float, HIPContext>(
          1, scale_ / h_total_weight,
          avg_loss_data, avg_loss_data, &context_);
    }
  }
  return true;
}


template<>
bool SoftmaxWithLossGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);  // Logits
  auto& T = Input(1);  // Labels / targets
  // Input(2) is weights, if given
  auto& P = Input(InputSize() - 2);  // Probabilities from softmax
  auto& d_avg_loss = Input(InputSize() - 1); // Gradient w.r.t. avg loss
  const float* weights = (InputSize() > 4 ? Input(2).data<float>() : NULL);

  auto* dX = Output(0);
  int N = X.dim32(0);
  int D = X.dim32(1);
  dX->ResizeLike(X);
  total_weight_ptr_.Resize(1);

  if (!spatial_mode_) {
    DCHECK_EQ(X.ndim(), 2);
    DCHECK(
        (T.ndim() == 1) || (T.ndim() == 2 && T.dim32(1) == 1) ||
        (T.ndim() == 2 && T.dim32(0) == N && T.dim32(1) == D));
    DCHECK_EQ(T.dim32(0), N);

    // Subtract 1 from labeled positions
    if (!label_prob_mode_) {
      if (weights == nullptr) {
        // Copy softmax probabilities into dX
        context_.Copy<float, HIPContext, HIPContext>(
            P.size(), P.data<float>(), dX->mutable_data<float>());
        LabelCrossEntropyGradientKernel<<<
            CAFFE_GET_BLOCKS(N),
            CAFFE_CUDA_NUM_THREADS,
            0,
            context_.hip_stream()>>>(
            N, D, P.data<float>(), T.data<int>(), dX->mutable_data<float>());
      } else {
        // Weighted version gets the Pdata values internally
        LabelCrossEntropyGradientKernelWeighted<<<
            CAFFE_GET_BLOCKS(N * D),
            CAFFE_CUDA_NUM_THREADS,
            0,
            context_.hip_stream()>>>(
            N,
            D,
            P.data<float>(),
            T.data<int>(),
            dX->mutable_data<float>(),
            weights);
      }
    } else {
      ProbCrossEntropyGradientKernel<<<
          CAFFE_GET_BLOCKS(N),
          CAFFE_CUDA_NUM_THREADS,
          0,
          context_.hip_stream()>>>(
          N,
          D,
          P.data<float>(),
          T.data<float>(),
          dX->mutable_data<float>(),
          weights);
    }
    float total_weight = N;
    if (weights) {
      // Sum weights
      math::Sum<float, HIPContext>(
        N, weights, total_weight_ptr_.mutable_data<float>(), &context_);
      hipMemcpyAsync(&total_weight, total_weight_ptr_.data<float>(),
        sizeof(float), hipMemcpyDeviceToHost, context_.hip_stream());
    }

    // Scale by d_avg_loss / N
    if (total_weight > 0) {
      math::Scale<float, HIPContext>(
          dX->size(),
          scale_ / total_weight,
          dX->data<float>(),
          dX->mutable_data<float>(),
          &context_);
    }
    math::Scale<float, HIPContext>(
        dX->size(), d_avg_loss.data<float>(), dX->data<float>(),
        dX->mutable_data<float>(), &context_);
  } else {
    // Spatial mode, compute softmax for each x, y location
    DCHECK_EQ(X.ndim(), 4);
    DCHECK_EQ(T.ndim(), 3);

    int H = X.dim32(2);
    int W = X.dim32(3);
    dX->ResizeLike(X);
    if (weights_.size() != N * W * H) {
      weights_.Resize(N * W * H);
    }

    const float* Pdata = P.data<float>();
    float* dX_data = dX->mutable_data<float>();
    const int* label_data = T.data<int>();
    const float* d_avg_loss_data = d_avg_loss.data<float>();

    // Copy softmax probabilities into dX. All but the neuron
    // corresponding to the correct label has gradient equaling e(x_j)
    // which is the probability under softmax.
    context_.Copy<float, HIPContext, HIPContext>(P.size(), Pdata, dX_data);

    math::Set<float, HIPContext>(
      1, 0.0f, total_weight_ptr_.mutable_data<float>(), &context_);

    SpatialSoftmaxLossGradientKernel<<<CAFFE_GET_BLOCKS(N * W * H),
      CAFFE_CUDA_NUM_THREADS, 0, context_.hip_stream()>>>(
        N, D, W, H, label_data, weights, dX_data,
        weights_.mutable_data<float>());

    math::Sum<float, HIPContext>(
      weights_.size(), weights_.data<float>(),
      total_weight_ptr_.mutable_data<float>(), &context_);

    // Somewhat awkward scalar passing from device to host
    float h_total_weight;
    hipMemcpyAsync(&h_total_weight, total_weight_ptr_.data<float>(),
      sizeof(float), hipMemcpyDeviceToHost, context_.hip_stream());

    // Final scaling
    if (h_total_weight > 0) {
      math::Scale<float, HIPContext>(
          dX->size(),
          scale_ / h_total_weight,
          dX->data<float>(),
          dX->mutable_data<float>(),
          &context_);
    }
    math::Scale<float, HIPContext>(
        dX->size(),
        d_avg_loss.data<float>(),
        dX->data<float>(),
        dX->mutable_data<float>(),
        &context_);
  }
  return true;
}


namespace {
REGISTER_HIP_OPERATOR(SoftmaxWithLoss,
                       SoftmaxWithLossOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(SoftmaxWithLossGradient,
                       SoftmaxWithLossGradientOp<float, HIPContext>);
} // namespace
} // namespace caffe2
