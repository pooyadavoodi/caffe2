#include "hip/hip_runtime.h"
// Implements the math functions for CPU.

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/system/cuda/detail/par.h>
#include <thrust/version.h>

#include "caffe2/utils/math.h"
#include "caffe2/core/context_gpu.h"

#if THRUST_VERSION >= 100800
#define THRUST_SUPPORTS_PER_THREAD
#endif  // THRUST_VERSION >= 100800

namespace caffe2 {
namespace math {

#define DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(T, Funcname, function)             \
__global__                                                                     \
void _Kernel_##T##_##Funcname(const int N, const T* x, T* y) {                 \
  CUDA_1D_KERNEL_LOOP(i, N) {                                                  \
    y[i] = function(x[i]);                                                     \
  }                                                                            \
}                                                                              \
template <>                                                                    \
void Funcname<T, HIPContext>(                                                 \
    const int N, const T* x, T* y,                                             \
    HIPContext* context) {                                                    \
  _Kernel_##T##_##Funcname<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,      \
                                 0, context->hip_stream()>>>(                 \
      N, x, y);                                                                \
}

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Exp, expf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Exp, exp);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Log, logf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Log, log);

__device__ float cuda_sqrf(const float x) { return x * x; }
__device__ double cuda_sqr(const double x) { return x * x; }

DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(float, Sqr, cuda_sqrf);
DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION(double, Sqr, cuda_sqr);

#undef DELEGATE_SIMPLE_CUDA_UNARY_FUNCTION

#define DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(T, Funcname, expr)          \
  __global__ void _Kernel_##T##_##Funcname(                              \
      const int N, const T* a, const T* b, T* y) {                       \
    CUDA_1D_KERNEL_LOOP(i, N) {                                          \
      y[i] = a[i] expr b[i];                                             \
    }                                                                    \
  }                                                                      \
  template <>                                                            \
  void Funcname<T, HIPContext>(                                         \
      const int N, const T* a, const T* b, T* y, HIPContext* context) { \
    _Kernel_##T##_##Funcname<<<                                          \
        CAFFE_GET_BLOCKS(N),                                             \
        CAFFE_CUDA_NUM_THREADS,                                          \
        0,                                                               \
        context->hip_stream()>>>(N, a, b, y);                           \
  }

DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Add, +);
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Sub, -);
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Mul, *);
DELEGATE_SIMPLE_CUDA_BINARY_FUNCTION(float, Div, /);

// Caffe2 gemm provides a simpler interface to the gemm functions, with the
// limitation that the data has to be contiguous in memory.
template <>
void Gemm<float, HIPContext>(
    const CBLAS_TRANSPOSE TransA, const CBLAS_TRANSPOSE TransB,
    const int M, const int N, const int K, const float alpha, const float* A,
    const float* B, const float beta, float* C, HIPContext* context) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  int lda = (TransA == CblasNoTrans) ? K : M;
  int ldb = (TransB == CblasNoTrans) ? N : K;
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      N));
}

template <>
void GemmEx<float, HIPContext>(
    const CBLAS_TRANSPOSE TransA, const CBLAS_TRANSPOSE TransB,
    const int M, const int N, const int K, const float alpha, const float* A,
    const int lda, const float* B, const int ldb, const float beta, float* C,
    const int ldc, HIPContext* context) {
  // Note that cublas follows fortran order, so the order is different from
  // the cblas convention.
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  hipblasOperation_t cuTransB =
      (TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  CUBLAS_ENFORCE(hipblasSgemm(
      context->cublas_handle(),
      cuTransB,
      cuTransA,
      N,
      M,
      K,
      &alpha,
      B,
      ldb,
      A,
      lda,
      &beta,
      C,
      ldc));
}

template <>
void Gemv<float, HIPContext>(
    const CBLAS_TRANSPOSE TransA, const int M, const int N, const float alpha,
    const float* A, const float* x, const float beta, float* y,
    HIPContext* context) {
  hipblasOperation_t cuTransA =
      (TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  CUBLAS_ENFORCE(hipblasSgemv(
      context->cublas_handle(),
      cuTransA,
      N,
      M,
      &alpha,
      A,
      N,
      x,
      1,
      &beta,
      y,
      1));
}


namespace {
template <typename T>
__global__ void SetKernel(const int N, const T alpha, T* Y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    Y[i] = alpha;
  }
}
}  // namespace

#define CAFFE2_SPECIALIZED_CUDA_SET(T)                                         \
  template <>                                                                  \
  void Set<T, HIPContext>(const TIndex N, const T alpha, T *Y,                \
                              HIPContext* context) {                          \
    SetKernel<<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,                   \
                0, context->hip_stream()>>>(N, alpha, Y);                     \
  }

CAFFE2_SPECIALIZED_CUDA_SET(float);
CAFFE2_SPECIALIZED_CUDA_SET(double);
CAFFE2_SPECIALIZED_CUDA_SET(bool);
CAFFE2_SPECIALIZED_CUDA_SET(int8_t);
CAFFE2_SPECIALIZED_CUDA_SET(int16_t);
CAFFE2_SPECIALIZED_CUDA_SET(int);
CAFFE2_SPECIALIZED_CUDA_SET(int64_t);
CAFFE2_SPECIALIZED_CUDA_SET(char);
CAFFE2_SPECIALIZED_CUDA_SET(uint8_t);
CAFFE2_SPECIALIZED_CUDA_SET(uint16_t);
#undef CAFFE2_SPECIALIZED_CUDA_SET

namespace {
template <typename T>
__global__ void UniformShift(const int N, const T min, const T max,
                             T* x) {
  T scale = max - min;
  CUDA_1D_KERNEL_LOOP(i, N) {
    x[i] = x[i] * scale + min;
  }
}

__global__ void UniformIntFit(const int N, const int min, const int max,
                              unsigned int* x) {
  int* x_int = reinterpret_cast<int*>(x);
  int range = (max - min + 1);
  CUDA_1D_KERNEL_LOOP(i, N) {
    x_int[i] = min + static_cast<int>(x[i] % range);
  }
}
}  // namespace

template <>
void RandUniform<float, HIPContext>(
    const int n, const float min, const float max, float* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerateUniform(context->curand_generator(), r, n));
  UniformShift<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                        0, context->hip_stream()>>>(n, min, max, r);
}

template <>
void RandUniform<double, HIPContext>(
    const int n, const double min, const double max, double* r,
    HIPContext* context) {
  CURAND_ENFORCE(
      hiprandGenerateUniformDouble(context->curand_generator(), r, n));
  UniformShift<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                         0, context->hip_stream()>>>(n, min, max, r);
}

template <>
void RandUniform<int, HIPContext>(
    const int n, const int min, const int max, int* r,
    HIPContext* context) {
  CURAND_ENFORCE(hiprandGenerate(
      context->curand_generator(), reinterpret_cast<unsigned int*>(r), n));
  UniformIntFit<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                  0, context->hip_stream()>>>(
      n, min, max, reinterpret_cast<unsigned int*>(r));
}

template <typename T>
int HandleOddLengthRandGaussian(
    const int n,
    const T mean,
    const T std,
    T* r,
    HIPContext* context) {
  if (n % 2 == 1) {
    std::default_random_engine generator;
    std::normal_distribution<T> distribution(mean, std);
    const T random_value = distribution(generator);
    math::Set<T, HIPContext>(
        1, random_value, r + sizeof(T) * (n - 1), context);
    return n - 1;
  }
  return n;
}

template <>
void RandGaussian<float, HIPContext>(
    const int n, const float mean, const float std, float* r,
    HIPContext* context) {
  // If n is odd, we add a random Gaussian value at the end manually
  // and generate n-1 random values using hiprandGenerateNormal.
  // hiprandGenerateNormal requires n to be even.
  const int even_n =
      HandleOddLengthRandGaussian<float>(n, mean, std, r, context);
  CURAND_ENFORCE(
      hiprandGenerateNormal(context->curand_generator(), r, even_n, mean, std));
}

template <>
void RandGaussian<double, HIPContext>(
    const int n, const double mean, const double std, double* r,
    HIPContext* context) {
  const int even_n =
      HandleOddLengthRandGaussian<double>(n, mean, std, r, context);
  CURAND_ENFORCE(hiprandGenerateNormalDouble(
      context->curand_generator(), r, even_n, mean, std));
}


template<>
void Dot<float, HIPContext>(
    const int n, const float* a, const float* b, float* y,
    HIPContext* context) {
  float result;
  CUBLAS_ENFORCE(hipblasSdot(context->cublas_handle(), n, a, 1, b, 1, &result));
  context->Copy<float, CPUContext, HIPContext>(1, &result, y);
}

template<>
void Dot<double, HIPContext>(
    const int n, const double* a, const double* b, double* y,
    HIPContext* context) {
  double result;
  CUBLAS_ENFORCE(hipblasDdot(context->cublas_handle(), n, a, 1, b, 1, y));
  context->Copy<double, CPUContext, HIPContext>(1, &result, y);
}

// A previous version of caffe2 used Thrust but it turns out that thrust
// reduction has an implicit scratch space allocation and deallocation, which
// may interfere with NCCL and create a deadlock. Hence we are using a custom
// reduction here.
#define SUM_KERNEL_NTHREADS 128
template <typename T>
__global__ void SumKernel(const int N, const T* X, T* Y) {
  const int idx = threadIdx.x;
  __shared__ T reduction_buffer[SUM_KERNEL_NTHREADS];

  reduction_buffer[idx] = 0;

  // A multilevel reduction.
  // N -> 128
  for (int i = idx; i < N; i += SUM_KERNEL_NTHREADS) {
    reduction_buffer[idx] += X[i];
  }
  __syncthreads();
  // 128 -> 32
  if (idx < 32) {
    reduction_buffer[idx] +=
        reduction_buffer[idx + 32] +
        reduction_buffer[idx + 64] +
        reduction_buffer[idx + 96];
  }
  __syncthreads();
  // 32 -> 1
  if (idx == 0) {
    float tmp = 0;
    for (int i = 0; i < 32; ++i) {
      tmp += reduction_buffer[i];
    }
    *Y = tmp;
  }
}

#define CAFFE2_MATH_SUM_FUNC(T)                                                \
template<>                                                                     \
void Sum<T, HIPContext>(const int N, const T* x, T* y, HIPContext* context) {\
  SumKernel<<<1, SUM_KERNEL_NTHREADS, 0, context->hip_stream()>>>(N, x, y);   \
}

CAFFE2_MATH_SUM_FUNC(float)
CAFFE2_MATH_SUM_FUNC(double)
#undef CAFFE2_MATH_SUM_FUNC

namespace {
template <typename T>
__global__ void SelectKernel(
    const int N, const int D, const T* x, const int* idx, T* y) {
  CUDA_1D_KERNEL_LOOP(i, N) {
    y[i] = x[i * D + idx[i]];
  }
}
}  // namespace

template <>
void Select<float, HIPContext>(
      const int N, const int D, const float* x, const int* idx, float* y,
      HIPContext* context) {
  SelectKernel<float><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS,
                        0, context->hip_stream()>>>(N, D, x, idx, y);
}

namespace {
template <typename T>
__global__ void ScaleKernel(
    const int n, const T alpha, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = x[i] * alpha;
  }
}

template <typename T>
__global__ void ScaleKernelDeviceAlpha(
    const int n, const T* alpha, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = x[i] * (*alpha);
  }
}

template <typename T>
__global__ void PowKernel(const int n, const T* x, const T exponent, T* y) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    y[i] = powf(x[i], exponent);
  }
}
}  // namespace

template <>
void Powx<float, HIPContext>(
    const int N,
    const float* a,
    const float b,
    float* y,
    HIPContext* context) {
  PowKernel<<<
      CAFFE_GET_BLOCKS(N),
      CAFFE_CUDA_NUM_THREADS,
      0,
      context->hip_stream()>>>(N, a, b, y);
}

template <>
void Scale<float, HIPContext>(
    const int n,
    const float alpha,
    const float* x,
    float* y,
    HIPContext* context) {
  ScaleKernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, alpha, x, y);
}

template <>
void Scale<double, HIPContext>(
    const int n, const double alpha, const double *x, double* y,
    HIPContext* context) {
  ScaleKernel<double><<<
      CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          n, alpha, x, y);
}

template <>
void Scale<float, HIPContext>(
    const int n, const float* alpha, const float *x, float* y,
    HIPContext* context) {
  ScaleKernelDeviceAlpha<float><<<
      CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS, 0, context->hip_stream()>>>(
          n, alpha, x, y);
}

template <>
void Scale<double, HIPContext>(
    const int n, const double* alpha, const double *x, double* y,
    HIPContext* context) {
  ScaleKernelDeviceAlpha<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, alpha, x, y);
}

template <>
void Axpy<float, HIPContext>(
    const int N,
    const float alpha,
    const float* X,
    float* Y,
    HIPContext* context) {
  CUBLAS_ENFORCE(hipblasSaxpy(context->cublas_handle(), N, &alpha, X, 1, Y, 1));
}

template <>
void Axpy<double, HIPContext>(
    const int N,
    const double alpha,
    const double* X,
    double* Y,
    HIPContext* context) {
  CUBLAS_ENFORCE(hipblasDaxpy(context->cublas_handle(), N, &alpha, X, 1, Y, 1));
}

namespace {
template <typename T>
__global__ void AxpyKernel(const int n, const T* a, const T* x, T* y) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    y[index] += x[index] * (*a);
  }
}
}  // namespace

template <>
void Axpy<float, HIPContext>(
    const int n, const float* alpha, const float* X,
    float* Y, HIPContext* context) {
  AxpyKernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, alpha, X, Y);
}

template <>
void Axpy<double, HIPContext>(
    const int n, const double* alpha, const double* X,
    double* Y, HIPContext* context) {
  AxpyKernel<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, alpha, X, Y);
}


namespace {
template <typename T>
__global__ void AxpbyKernel(const int n, const T a, const T* x,
                             const T b, T* y) {
  CUDA_1D_KERNEL_LOOP(index, n) {
    y[index] = x[index] * a + y[index] * b;
  }
}
}  // namespace

template <>
void Axpby<float, HIPContext>(
    const int n, const float a, const float* x, const float b, float* y,
    HIPContext* context) {
  AxpbyKernel<float><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                       0, context->hip_stream()>>>(n, a, x, b, y);
}

template <>
void Axpby<double, HIPContext>(
    const int n, const double a, const double* x, const double b, double* y,
    HIPContext* context) {
  AxpbyKernel<double><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS,
                        0, context->hip_stream()>>>(n, a, x, b, y);
}

namespace {

template <typename T>
__global__ void im2col_gpu_kernel_nchw(const int n, const T* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    T* data_col) {

  CUDA_1D_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w;
    int h_in = h_out * stride_h - pad_t;
    int w_in = w_out * stride_w - pad_l;
    T* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const T* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h = h_in + i * dilation_h;
        int w = w_in + j * dilation_w;
        *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename T>
__global__ void im2col_gpu_kernel_nhwc(const int n, const T* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int width_col, const int channels,
    T* data_col) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    int channel_in = index % channels;
    int w_out = index / channels % width_col;
    int h_out = index / channels / width_col;
    int h_in = h_out * stride_h - pad_t;
    int w_in = w_out * stride_w - pad_l;
    T* local_data_col = data_col +
        ((h_out * width_col) + w_out) * channels * kernel_h * kernel_w
        + channel_in;
    for (int i = 0; i < dkernel_h; i += dilation_h) {
      int h = h_in + i;
      for (int j = 0; j < dkernel_w; j += dilation_w) {
        int w = w_in + j;
        *local_data_col = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im[(h * width + w) * channels + channel_in] : 0;
        local_data_col += channels;
      }
    }
  }
}

template <typename T>
__global__ void col2im_gpu_kernel_nchw(const int n, const T* data_col,
    const int height, const int width,
    const int patch_h, const int patch_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    T* data_im) {

  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    int w = index % width + pad_l;
    int h = (index / width) % height + pad_t;
    int c = index / (width * height);

    // compute the start and end of the output
    int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = (h - h_col * stride_h);
        int w_k = (w - w_col * stride_w);
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int data_col_index =
              (((c * patch_h + h_k) * patch_w + w_k) * height_col + h_col) *
                  width_col +
              w_col;
          val += data_col[data_col_index];
        }
      }
    }
    data_im[index] = val;
  }
}

template <typename T>
__global__ void col2im_gpu_kernel_nhwc(const int n, const T* data_col,
    const int width, const int channels,
    const int patch_h, const int patch_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    T* data_im) {

  const int dpatch_h = dilation_h * (patch_h - 1) + 1;
  const int dpatch_w = dilation_w * (patch_w - 1) + 1;

  CUDA_1D_KERNEL_LOOP(index, n) {
    T val = 0;
    int c = index % channels;
    int w = index / channels % width + pad_l;
    int h = index / channels / width + pad_t;
    // compute the start and end of the output
    int w_col_start = (w < dpatch_w) ? 0 : (w - dpatch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < dpatch_h) ? 0 : (h - dpatch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);
    int channels_col = patch_h * patch_w * channels;

    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        int h_k = h - h_col * stride_h;
        int w_k = w - w_col * stride_w;
        if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
          h_k /= dilation_h;
          w_k /= dilation_w;
          int c_col = (h_k * patch_w + w_k) * channels + c;
          val += data_col[(h_col * width_col + w_col) * channels_col + c_col];
        }
      }
    }
    data_im[index] = val;
  }
}

// Ported from caffe1
template <typename T, int num_axes>
__global__ void im2col_nd_gpu_kernel(
    const int n,
    const T* data_im,
    const int* im_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* pad,
    const int* stride,
    const int* dilation,
    T* data_col) {
  int d_temp[num_axes]; // NOLINT(runtime/arrays)
  int d_iter[num_axes]; // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }
  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  int i;
  CUDA_1D_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_in = index;
    int channel_out = 1;
    for (i = num_axes - 1; i >= 0; --i) {
      d_temp[i] = channel_in % shared_col_shape[i + 1];
      channel_in /= shared_col_shape[i + 1];
      channel_out *= shared_kernel_shape[i];
    }
    channel_out *= channel_in;
    int data_col_inc = 1;
    for (i = 0; i < num_axes; ++i) {
      channel_out *= shared_col_shape[i + 1];
      channel_out += d_temp[i];
      d_temp[i] = d_temp[i] * shared_stride[i] - shared_pad[i];
      channel_in *= shared_im_shape[i + 1];
      channel_in += d_temp[i];
      data_col_inc *= shared_col_shape[i + 1];
      d_iter[i] = 0;
    }
    T* data_col_ptr = data_col + channel_out;
    const T* data_im_ptr = data_im + channel_in;
    bool incremented;
    do {
      bool in_range = true;
      for (i = 0; i < num_axes; ++i) {
        const int d_iter_im = d_iter[i] * shared_dilation[i] + d_temp[i];
        in_range &= d_iter_im >= 0 && d_iter_im < shared_im_shape[i + 1];
        if (!in_range) {
          break;
        }
      }
      if (in_range) {
        int data_im_offset = d_iter[0] * shared_dilation[0];
        for (i = 1; i < num_axes; ++i) {
          data_im_offset *= shared_im_shape[i + 1];
          data_im_offset += d_iter[i] * shared_dilation[i];
        }
        *data_col_ptr = data_im_ptr[data_im_offset];
      } else {
        *data_col_ptr = 0;
      }
      data_col_ptr += data_col_inc;
      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        const int d_max = shared_kernel_shape[i];
        if (d_iter[i] == d_max - 1) {
          d_iter[i] = 0;
        } else { // d_iter[i] < d_max - 1
          ++d_iter[i];
          incremented = true;
          break;
        }
      } // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented); // do
  } // CUDA_KERNEL_LOOP(index, n)
}

template <typename T, int num_axes>
__global__ void col2im_nd_gpu_kernel(
    const int n,
    const T* data_col,
    const int* im_shape,
    const int* col_shape,
    const int* kernel_shape,
    const int* pad,
    const int* stride,
    const int* dilation,
    T* data_im) {
  int d_im[num_axes]; // NOLINT(runtime/arrays)
  int d_col_iter[num_axes]; // NOLINT(runtime/arrays)
  int d_col_start[num_axes]; // NOLINT(runtime/arrays)
  int d_col_end[num_axes]; // NOLINT(runtime/arrays)

  __shared__ int shared_dilation[num_axes];
  __shared__ int shared_kernel_shape[num_axes];
  __shared__ int shared_pad[num_axes];
  __shared__ int shared_stride[num_axes];
  __shared__ int shared_col_shape[num_axes + 1];
  __shared__ int shared_im_shape[num_axes + 1];

  if (threadIdx.x < num_axes) {
    shared_dilation[threadIdx.x] = dilation[threadIdx.x];
    shared_kernel_shape[threadIdx.x] = kernel_shape[threadIdx.x];
    shared_pad[threadIdx.x] = pad[threadIdx.x];
    shared_stride[threadIdx.x] = stride[threadIdx.x];
  }

  if (threadIdx.x < num_axes + 1) {
    shared_col_shape[threadIdx.x] = col_shape[threadIdx.x];
    shared_im_shape[threadIdx.x] = im_shape[threadIdx.x];
  }
  __syncthreads();

  CUDA_1D_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int c_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = c_im % shared_im_shape[i + 1] + shared_pad[i];
      c_im /= shared_im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      const int kernel_extent =
          shared_dilation[i] * (shared_kernel_shape[i] - 1) + 1;
      d_col_start[i] = d_col_iter[i] = (d_im[i] < kernel_extent)
          ? 0
          : (d_im[i] - kernel_extent) / shared_stride[i] + 1;
      d_col_end[i] =
          min(d_im[i] / shared_stride[i] + 1, shared_col_shape[i + 1]);
      if (d_col_start[i] >= d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index] = 0;
        done = true;
        break; // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue; // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    T val = 0;
    bool incremented = true;
    bool skip = false;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int kernel_shape_prod = 1;
      int kernel_index;
      for (int i = num_axes - 1; i >= 0; --i) {
        kernel_index = d_im[i] - d_col_iter[i] * shared_stride[i];
        if (kernel_index % shared_dilation[i]) {
          skip = true;
          break;
        } else {
          kernel_index /= shared_dilation[i];
          final_offset += kernel_index * kernel_shape_prod;
          kernel_shape_prod *= shared_kernel_shape[i];
        }
      }
      if (!skip) {
        final_offset += kernel_shape_prod * c_im;
        for (int i = 0; i < num_axes; ++i) {
          final_offset *= shared_col_shape[i + 1];
          final_offset += d_col_iter[i];
        }
        val += data_col[final_offset];
      }
      skip = false;
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        const int d_max = d_col_end[i];
        if (d_col_iter[i] == d_max - 1) {
          d_col_iter[i] = d_col_start[i];
        } else { // d_col_iter[i] < d_max - 1
          ++d_col_iter[i];
          incremented = true;
          break; // for (int i = num_axes - 1; i >= 0; --i)
        }
      } // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);
    data_im[index] = val;
  } // CUDA_KERNEL_LOOP(index, n)
}

}  // namespace

template <>
void Im2col<float, HIPContext, StorageOrder::NCHW>(
    const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_col, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel_nchw<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      dilation_h, dilation_w, pad_t, pad_l, stride_h, stride_w,
      height_col, width_col, data_col);
}

template <>
void Im2col<float, HIPContext, StorageOrder::NHWC>(
    const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_col, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  // We are going to launch height_col * width_col * channels kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = height_col * width_col * channels;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel_nhwc<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w,
      dilation_h, dilation_w, pad_t, pad_l, stride_h, stride_w,
      width_col, channels, data_col);
}


template <>
void Col2im<float, HIPContext, StorageOrder::NCHW>(
    const float* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_im, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  col2im_gpu_kernel_nchw<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_col, height, width, kernel_h, kernel_w,
      dilation_h, dilation_w,
      pad_t, pad_l, stride_h, stride_w,
      height_col, width_col, data_im);
}

template <>
void Col2im<float, HIPContext, StorageOrder::NHWC>(
    const float* data_col, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int dilation_h, const int dilation_w,
    const int pad_t, const int pad_l, const int pad_b, const int pad_r,
    const int stride_h,
    const int stride_w, float* data_im, HIPContext* context) {

  const int dkernel_h = dilation_h * (kernel_h - 1) + 1;
  const int dkernel_w = dilation_w * (kernel_w - 1) + 1;

  int height_col = (height + pad_t + pad_b - dkernel_h) / stride_h + 1;
  int width_col = (width + pad_l + pad_r - dkernel_w) / stride_w + 1;
  int num_kernels = height * width * channels;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  col2im_gpu_kernel_nhwc<float><<<CAFFE_GET_BLOCKS(num_kernels),
                                  CAFFE_CUDA_NUM_THREADS, 0,
                                  context->hip_stream()>>>(
      num_kernels, data_col, width, channels, kernel_h, kernel_w,
      dilation_h, dilation_w,
      pad_t, pad_l, stride_h, stride_w, height_col, width_col, data_im);
}

template <>
void Col2imNd<float, HIPContext, StorageOrder::NCHW>(
    const float* data_col,
    const int* img_shape,
    const int* col_shape,
    const int img_size,
    const int col_size,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const int N,
    float* data_img,
    HIPContext* context) {
  CAFFE_ENFORCE_LT(
      N, CAFFE_CUDA_NUM_THREADS, "num_axes should be smaller than block size.");

#define COL2IM_ND_KERNEL(n)                                                   \
  col2im_nd_gpu_kernel<float, n> /* NOLINT_NEXT_LINE(whitespace/operators) */ \
      <<<CAFFE_GET_BLOCKS(img_size),                                          \
         CAFFE_CUDA_NUM_THREADS,                                              \
         0,                                                                   \
         context->hip_stream()>>>(                                           \
          img_size,                                                           \
          data_col,                                                           \
          img_shape,                                                          \
          col_shape,                                                          \
          kernel_shape,                                                       \
          pad,                                                                \
          stride,                                                             \
          dilation,                                                           \
          data_img)

  switch (N) {
    case 1:
      COL2IM_ND_KERNEL(1);
      break;
    case 2:
      COL2IM_ND_KERNEL(2);
      break;
    case 3:
      COL2IM_ND_KERNEL(3);
      break;
    case 4:
      COL2IM_ND_KERNEL(4);
      break;
    case 5:
      COL2IM_ND_KERNEL(5);
      break;
    default:
      CAFFE_THROW(
          "Col2imNd does not support computation with ", N, " spatial axes");
  }
}

template <>
void Im2colNd<float, HIPContext, StorageOrder::NCHW>(
    const float* data_img,
    const int* img_shape,
    const int* col_shape,
    const int img_size,
    const int col_size,
    const int* kernel_shape,
    const int* stride,
    const int* dilation,
    const int* pad,
    const int N,
    float* data_col,
    HIPContext* context,
    bool /*accumlate_output*/) {
  CAFFE_ENFORCE_LT(
      N, CAFFE_CUDA_NUM_THREADS, "num_axes should be smaller than block size.");

#define IM2COL_ND_KERNEL(n)                                                   \
  im2col_nd_gpu_kernel<float, n> /* NOLINT_NEXT_LINE(whitespace/operators) */ \
      <<<CAFFE_GET_BLOCKS(col_size),                                          \
         CAFFE_CUDA_NUM_THREADS,                                              \
         0,                                                                   \
         context->hip_stream()>>>(                                           \
          col_size,                                                           \
          data_img,                                                           \
          img_shape,                                                          \
          col_shape,                                                          \
          kernel_shape,                                                       \
          pad,                                                                \
          stride,                                                             \
          dilation,                                                           \
          data_col)

  switch (N) {
    case 1:
      IM2COL_ND_KERNEL(1);
      break;
    case 2:
      IM2COL_ND_KERNEL(2);
      break;
    case 3:
      IM2COL_ND_KERNEL(3);
      break;
    case 4:
      IM2COL_ND_KERNEL(4);
    case 5:
      IM2COL_ND_KERNEL(5);
      break;
    default:
      CAFFE_THROW(
          "Im2colNd does not support computation with ", N, " spatial axes");
  }
}

template <>
void CopyMatrix<HIPContext>(
    const size_t itemsize,
    const int M,
    const int N,
    const void* A,
    const int lda,
    void* B,
    const int ldb,
    HIPContext* context) {
  hipMemcpy2DAsync(B, ldb * itemsize, A, lda * itemsize, N * itemsize, M,
                    hipMemcpyDeviceToDevice, context->hip_stream());
}

}  // namespace math
}  // namespace caffe2
